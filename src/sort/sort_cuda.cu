#include "hip/hip_runtime.h"
#include <float.h>

#include "sort_cuda.cuh"

// __host__ static size_t GetSmallestPowerOfTwoLargerThan(size_t n) {
//   size_t k = 1;
//   while (k < n) {
//     k = k << 1;
//   }
//   return k;
// }

__host__ hipError_t InitialCuda(int device) {
  // 初始化CUDA设备, 线程级别!
  hipError_t cudaStatus;

  // 清除遗留错误
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "\n[Error] last execution failed: %s!\n",
            hipGetErrorString(cudaStatus));
  }

  // 确定CUDA设备, 默认只选中第一个设备
  cudaStatus = hipSetDevice(device);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr,
            "\n[Error] hipSetDevice failed!  Do you have a CUDA-capable GPU "
            "installed?\n");
  }

  return cudaStatus;
}

__host__ hipError_t ReleaseCuda(void) {
  // 重置CUDA设备, 进程级别!
  hipError_t cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "\n[Error] hipDeviceReset failed!\n");
  }

  return cudaStatus;
}
__global__ void BitonicSortKernal(float* data, size_t step, size_t len,
                                  bool dir) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;

  float tmp;
  size_t i = tid % step;
  if (i < step / 2) {
    if ((data[tid] < data[tid + step - 2 * i - 1]) ^ dir) {
      tmp = data[tid];
      data[tid] = data[tid + step - 2 * i - 1];
      data[tid + step - 2 * i - 1] = tmp;
    }
  }

  __syncthreads();
}

__global__ void BitonicMergeKernal(float* data, size_t step, size_t len,
                                   bool dir) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;

  float tmp;
  size_t i = tid % (2 * step);
  if (i < step) {
    if ((data[tid] < data[tid + step]) ^ dir) {
      tmp = data[tid];
      data[tid] = data[tid + step];
      data[tid + step] = tmp;
    }
  }
  __syncthreads();
}

__global__ void InitKernal(float* ptr, size_t len, bool dir) {
  size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid >= len) return;
  ptr[tid] = dir ? FLT_MAX : -FLT_MAX;
}

__host__ void SortWithCuda(float* data_host, size_t len, bool dir) {
  int block_size = 1024;
  dim3 block(block_size, 1);
  size_t num = 1;
  while (num < len) {
    num <<= 1;
  }

  float* data_dev = NULL;

  // may cause large memory waste
  hipMalloc((void**)&data_dev, num * sizeof(float));

  // init data according to dir
  dim3 init_grid((num - len - 1) / block.x + 1, 1);
  hipDeviceSynchronize();
  InitKernal<<<init_grid, block>>>(data_dev + len, num - len, dir);
  hipDeviceSynchronize();

  hipMemcpy(data_dev, data_host, len * sizeof(float), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  dim3 grid((num - 1) / block.x + 1, 1);
  for (size_t step = 1; step <= num; step <<= 1) {
    BitonicSortKernal<<<grid, block>>>(data_dev, step, num, dir);
    for (size_t s = step / 4; s > 0; s /= 2)
      BitonicMergeKernal<<<grid, block>>>(data_dev, s, num, dir);
  }
  hipDeviceSynchronize();
  hipMemcpy(data_host, data_dev, len * sizeof(float), hipMemcpyDeviceToHost);
}