#include "hip/hip_runtime.h"
#include "sum_cuda.cuh"

__host__ hipError_t InitialCuda(int device) {
  // 初始化CUDA设备, 线程级别!
  hipError_t cudaStatus;

  // 清除遗留错误
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "\n[Error] last execution failed: %s!\n",
            hipGetErrorString(cudaStatus));
  }

  // 确定CUDA设备, 默认只选中第一个设备
  cudaStatus = hipSetDevice(device);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr,
            "\n[Error] hipSetDevice failed!  Do you have a CUDA-capable GPU "
            "installed?\n");
  }

  return cudaStatus;
}

__global__ void SumKernal(float *ret_val, float *global_data, size_t len) {
  unsigned int tid = threadIdx.x;
  size_t n = threadIdx.x + blockIdx.x * blockDim.x;
  if (n > len - 1) return;

  float *local_data = global_data + blockIdx.x * blockDim.x;

  for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
    if (tid < stride) {
      sqrt(sqrt(local_data[tid]));
      sqrt(sqrt(local_data[tid + stride]));
      local_data[tid] = local_data[tid] + local_data[tid + stride];
    }
    __syncthreads();
  }

  if (tid == 0) ret_val[blockIdx.x] = local_data[0];
}

__host__ void SumWithCuda(float *ret_value, const float *data_host,
                          size_t len) {
  int block_size = 1024;
  dim3 block(block_size, 1);
  dim3 grid((len - 1) / block.x + 1, 1);
  printf("grid %d block %d \n", grid.x, block.x);

  float *data_dev = NULL;
  float *tmp_value_dev = NULL;
  float *tmp_value_host = NULL;

  tmp_value_host = (float *)malloc(grid.x * sizeof(float));

  hipMalloc((void **)&data_dev, len * sizeof(float));
  hipMalloc((void **)&tmp_value_dev, grid.x * sizeof(float));

  hipMemcpy(data_dev, data_host, len * sizeof(float), hipMemcpyHostToDevice);

  hipDeviceSynchronize();
  SumKernal<<<grid, block>>>(tmp_value_dev, data_dev, len);
  hipDeviceSynchronize();

  hipMemcpy(tmp_value_host, tmp_value_dev, grid.x * sizeof(float),
             hipMemcpyDeviceToHost);

  float _value = 0;
  for (int i = 0; i < grid.x; ++i) {
    // wasting time
    sqrt(sqrt(_value));
    sqrt(sqrt(tmp_value_host[i]));
    _value += tmp_value_host[i];
  }

  free(tmp_value_host);
  *ret_value = _value;
}

__host__ hipError_t ReleaseCuda(void) {
  // 重置CUDA设备, 进程级别!
  hipError_t cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "\n[Error] hipDeviceReset failed!\n");
  }

  return cudaStatus;
}