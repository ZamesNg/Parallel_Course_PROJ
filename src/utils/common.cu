#include "common.cuh"

__host__ hipError_t InitialCuda(int device) {
  // 初始化CUDA设备, 线程级别!
  hipError_t cudaStatus;

  // 清除遗留错误
  cudaStatus = hipGetLastError();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "\n[Error] last execution failed: %s!\n",
            hipGetErrorString(cudaStatus));
  }

  // 确定CUDA设备, 默认只选中第一个设备
  cudaStatus = hipSetDevice(device);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr,
            "\n[Error] hipSetDevice failed!  Do you have a CUDA-capable GPU "
            "installed?\n");
  }

  return cudaStatus;
}

__host__ hipError_t ReleaseCuda(void) {
  // 重置CUDA设备, 进程级别!
  hipError_t cudaStatus = hipDeviceReset();
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "\n[Error] hipDeviceReset failed!\n");
  }

  return cudaStatus;
}